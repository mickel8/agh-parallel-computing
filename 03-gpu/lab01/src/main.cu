#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include "timings/helper_timer.h"

__global__ void add(int *a, int *b, int *c, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size) {
        c[tid] = a[tid] + b[tid];
    }
}

void add_cpu(int *a, int *b, int *c, int size) {
    for (int i = 0; i < size; i++) {
        c[i] = a[i] + b[i];
    }
}

bool check(int *a, int *b, int size) {
    for (int i = 0; i < size; i++) {
        if (a[i] != b[i]) {
            return false;
        }
    }
    return true;
}

int main(void) {

    int size, threads, blocks;
    printf("enter vector size: ");
    scanf("%d", &size);
    printf("enter threads per block: ");
    scanf("%d", &threads);
    printf("enter blocks per grid: ");
    scanf("%d", &blocks);
    
    int *a = (int*)calloc(size, sizeof(int));
    int *b = (int*)calloc(size, sizeof(int));
    int *c = (int*)calloc(size, sizeof(int));
    int *c_cpu = (int *)calloc(size, sizeof(int));

    int *dev_a, *dev_b, *dev_c;
    hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));
    hipMalloc((void**)&dev_c, size * sizeof(int));
    
    for (int i = 0; i < size; i++) {
        a[i] = i;
        b[i] = i * 2;
    }
    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_c, c, size * sizeof(int), hipMemcpyHostToDevice);
    
    StopWatchInterface *timer=NULL;
    sdkCreateTimer(&timer);
    sdkResetTimer(&timer);
    sdkStartTimer(&timer);
    add <<<blocks, threads>>> (dev_a, dev_b, dev_c, size);
    hipDeviceSynchronize();
    sdkStopTimer(&timer);
    float time = sdkGetTimerValue(&timer);
    sdkDeleteTimer(&timer);

    hipMemcpy(c, dev_c, size*sizeof(int), hipMemcpyDeviceToHost);
    // for (int i = 0; i < size; i++) {
    //     printf("%d+%d=%d\n", a[i], b[i], c[i]);
    // }
    printf("time for the kernel %f (ms) \n", time);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    
//    clock_t t;
//    t = clock();
//    add_cpu(a, b, c_cpu, size);
//    t = clock() - t;
//    printf("time for the cpu fun %f (ms) \n", ((double)t)/CLOCKS_PER_SEC * 1000);
//
//    if (check(c, c_cpu, size)) {
//        printf("Results are the same\n");
//    } else {
//        printf("Results are not the same\n");
//    }

    free(a);
    free(b);
    free(c);
    free(c_cpu);

    return 0;
}
